#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <sstream>
#include <string>
#include <algorithm>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <cfloat>
#include <cmath>

#define CHECK_CUDA(call) { hipError_t err = call; if (err != hipSuccess) { printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); exit(1); } }

struct Vec3f {
    float x, y, z;
    __host__ __device__ Vec3f() : x(0), y(0), z(0) {}
    __host__ __device__ Vec3f(float x, float y, float z) : x(x), y(y), z(z) {}
    __host__ __device__ Vec3f operator+(const Vec3f& v) const { return Vec3f(x + v.x, y + v.y, z + v.z); }
    __host__ __device__ Vec3f operator-(const Vec3f& v) const { return Vec3f(x - v.x, y - v.y, z - v.z); }
    __host__ __device__ Vec3f operator*(float f) const { return Vec3f(x * f, y * f, z * f); }
    __host__ __device__ float dot(const Vec3f& v) const { return x * v.x + y * v.y + z * v.z; }
    __host__ __device__ Vec3f cross(const Vec3f& v) const { return Vec3f(y * v.z - z * v.y, z * v.x - x * v.z, x * v.y - y * v.x); }
    __host__ __device__ Vec3f normalize() const { float l = sqrt(x * x + y * y + z * z); return Vec3f(x / l, y / l, z / l); }
    __host__ __device__ float& operator[](int i) { return i == 0 ? x : (i == 1 ? y : z); }
    __host__ __device__ const float& operator[](int i) const { return i == 0 ? x : (i == 1 ? y : z); }
};

struct Vec2f {
    float u, v;
    __host__ __device__ Vec2f() : u(0), v(0) {}
    __host__ __device__ Vec2f(float u, float v) : u(u), v(v) {}
    __host__ __device__ Vec2f operator*(float f) const { return Vec2f(u * f, v * f); }
    __host__ __device__ Vec2f operator+(const Vec2f& other) const { return Vec2f(u + other.u, v + other.v); }
};

struct Triangle {
    Vec3f v[3];
    Vec2f uv[3];
    Vec3f n[3];
};

struct Mat4f {
    float m[4][4];

    __host__ __device__ Mat4f() {
        for (int i = 0; i < 4; i++)
            for (int j = 0; j < 4; j++)
                m[i][j] = (i == j) ? 1.0f : 0.0f;
    }

    __host__ __device__ Vec3f transform(const Vec3f& v) const {
        float x = m[0][0] * v.x + m[0][1] * v.y + m[0][2] * v.z + m[0][3];
        float y = m[1][0] * v.x + m[1][1] * v.y + m[1][2] * v.z + m[1][3];
        float z = m[2][0] * v.x + m[2][1] * v.y + m[2][2] * v.z + m[2][3];
        float w = m[3][0] * v.x + m[3][1] * v.y + m[3][2] * v.z + m[3][3];
        return Vec3f(x/w, y/w, z/w);
    }

    __host__ __device__ Vec3f transformNormal(const Vec3f& n) const {
        float x = m[0][0] * n.x + m[0][1] * n.y + m[0][2] * n.z;
        float y = m[1][0] * n.x + m[1][1] * n.y + m[1][2] * n.z;
        float z = m[2][0] * n.x + m[2][1] * n.y + m[2][2] * n.z;
        return Vec3f(x, y, z).normalize();
    }
};

__global__ void rasterize_kernel(Triangle* triangles, int* triangle_counts, 
                                 unsigned char* textures, int* tex_widths, int* tex_heights, 
                                 unsigned char* output, float* zbuffer, 
                                 int width, int height, int num_objects) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int flipped_y = height - 1 - y;
    int idx = flipped_y * width + x;
    zbuffer[idx] = FLT_MAX;

    Vec3f color(0.2f, 0.2f, 0.2f);
    Vec3f light_dir = Vec3f(1, 1, 1).normalize();
    Vec3f P(x, flipped_y, 0);

    int triangle_offset = 0;
    int texture_offset = 0;

    for (int obj = 0; obj < num_objects; obj++) {
        for (int i = 0; i < triangle_counts[obj]; i++) {
            Triangle& tri = triangles[triangle_offset + i];
            
            Vec3f screen_coords[3];
            for (int j = 0; j < 3; j++) {
                Vec3f v = tri.v[j];
                screen_coords[j] = Vec3f((v.x + 1.0f) * width / 2.0f, (1.0f - v.y) * height / 2.0f, v.z);
            }

            // barycentric
            Vec3f s[2];
            for (int k = 2; k--; ) {
                s[k].x = screen_coords[2][k] - screen_coords[0][k];
                s[k].y = screen_coords[1][k] - screen_coords[0][k];
                s[k].z = screen_coords[0][k] - P[k];
            }
            Vec3f u = s[0].cross(s[1]);
            Vec3f bc_screen;
            if (std::abs(u.z) > 1e-2)
                bc_screen = Vec3f(1.f - (u.x + u.y) / u.z, u.y / u.z, u.x / u.z);
            else
                bc_screen = Vec3f(-1, 1, 1);

            if (bc_screen.x < 0 || bc_screen.y < 0 || bc_screen.z < 0) continue;

            float frag_depth = bc_screen.x * screen_coords[0].z + bc_screen.y * screen_coords[1].z + bc_screen.z * screen_coords[2].z;
            if (frag_depth < zbuffer[idx]) {
                zbuffer[idx] = frag_depth;

                Vec2f uv = tri.uv[0] * bc_screen.x + tri.uv[1] * bc_screen.y + tri.uv[2] * bc_screen.z;
                int tex_x = uv.u * tex_widths[obj];
                int tex_y = (1.0f - uv.v) * tex_heights[obj];

                int tex_idx = texture_offset + (tex_y * tex_widths[obj] + tex_x) * 3;
                Vec3f tex_color(textures[tex_idx] / 255.0f, textures[tex_idx + 1] / 255.0f, textures[tex_idx + 2] / 255.0f);

                Vec3f normal = (tri.n[0] * bc_screen.x + tri.n[1] * bc_screen.y + tri.n[2] * bc_screen.z).normalize();

                float diffuse = max(0.0f, normal.dot(light_dir));
                color = tex_color * (0.3f + 0.7f * diffuse);
            }
        }
        triangle_offset += triangle_counts[obj];
        texture_offset += tex_widths[obj] * tex_heights[obj] * 3;
    }

    output[idx * 3 + 0] = static_cast<unsigned char>(min(color.x * 255.0f, 255.0f));
    output[idx * 3 + 1] = static_cast<unsigned char>(min(color.y * 255.0f, 255.0f));
    output[idx * 3 + 2] = static_cast<unsigned char>(min(color.z * 255.0f, 255.0f));
}

void load_obj(const char* filename, std::vector<Triangle>& triangles) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        printf("Failed to open OBJ file: %s\n", filename);
        return;
    }

    std::vector<Vec3f> vertices, normals;
    std::vector<Vec2f> texcoords;
    std::string line, type;

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        iss >> type;

        if (type == "v") {
            Vec3f v;
            iss >> v.x >> v.y >> v.z;
            vertices.push_back(v);
        } else if (type == "vt") {
            Vec2f vt;
            iss >> vt.u >> vt.v;
            texcoords.push_back(vt);
        } else if (type == "vn") {
            Vec3f vn;
            iss >> vn.x >> vn.y >> vn.z;
            normals.push_back(vn);
        } else if (type == "f") {
            Triangle tri;
            for (int i = 0; i < 3; i++) {
                int v, vt, vn;
                char slash;
                iss >> v >> slash >> vt >> slash >> vn;
                tri.v[i] = vertices[v - 1];
                tri.uv[i] = texcoords[vt - 1];
                tri.n[i] = normals[vn - 1];
            }
            triangles.push_back(tri);
        }
    }
}

Mat4f create_model_matrix(float tx, float ty, float tz, float scale = 1.0f, float rotation = 0.0f) {
    Mat4f matrix;
    
    // Scale
    matrix.m[0][0] = matrix.m[1][1] = matrix.m[2][2] = scale;
    
    // Rotation (around Y-axis)
    float cos_r = cos(rotation);
    float sin_r = sin(rotation);
    matrix.m[0][0] = cos_r * scale;
    matrix.m[0][2] = -sin_r * scale;
    matrix.m[2][0] = sin_r * scale;
    matrix.m[2][2] = cos_r * scale;
    
    // Translation
    matrix.m[0][3] = tx;
    matrix.m[1][3] = ty;
    matrix.m[2][3] = tz;
    
    return matrix;
}

Mat4f create_perspective_matrix(float fov, float aspect, float near, float far) {
    Mat4f result;
    float tanHalfFov = tan(fov / 2.0f);
    
    result.m[0][0] = 1.0f / (aspect * tanHalfFov);
    result.m[1][1] = 1.0f / tanHalfFov;
    result.m[2][2] = -(far + near) / (far - near);
    result.m[2][3] = -2.0f * far * near / (far - near);
    result.m[3][2] = -1.0f;
    result.m[3][3] = 0.0f;
    
    return result;
}

int main() {
    const int width = 340, height = 280;
    const int num_objects = 2;
    
    // Load objects and textures
    std::vector<Triangle> triangles[num_objects];
    unsigned char* textures[num_objects];
    int tex_widths[num_objects], tex_heights[num_objects];
    
    load_obj("african_head.obj", triangles[0]);
    load_obj("drone.obj", triangles[1]);
    
    textures[0] = stbi_load("african_head_diffuse.tga", &tex_widths[0], &tex_heights[0], nullptr, 3);
    textures[1] = stbi_load("drone.png", &tex_widths[1], &tex_heights[1], nullptr, 3);
    
    // Prepare model matrices
    Mat4f model_matrices[num_objects] = {
        create_model_matrix(-1.0f, 0.0f, -3.0f, 1.0f, 3.14159f * 1.75f), // African head
        create_model_matrix(1.0f, 0.5f, -2.5f, 0.1f)  // Drone
    };

    // Prepare projection matrix
    Mat4f proj = create_perspective_matrix(3.14159f / 4.0f, (float)width / height, 0.1f, 100.0f);

    // Project vertices and transform normals directly in the triangles
    for (int i = 0; i < num_objects; i++) {
        for (auto& tri : triangles[i]) {
            for (int j = 0; j < 3; j++) {
                tri.v[j] = proj.transform(model_matrices[i].transform(tri.v[j]));
                tri.n[j] = model_matrices[i].transformNormal(tri.n[j]);
            }
        }
    }

    // Prepare GPU data
    Triangle* d_triangles;
    unsigned char* d_textures;
    int* d_triangle_counts, *d_tex_widths, *d_tex_heights;
    unsigned char* d_output;
    float* d_zbuffer;

    int total_triangles = triangles[0].size() + triangles[1].size();
    int total_texture_size = (tex_widths[0] * tex_heights[0] + tex_widths[1] * tex_heights[1]) * 3;

    // Allocate GPU memory
    CHECK_CUDA(hipMalloc(&d_triangles, total_triangles * sizeof(Triangle)));
    CHECK_CUDA(hipMalloc(&d_textures, total_texture_size * sizeof(unsigned char)));
    CHECK_CUDA(hipMalloc(&d_triangle_counts, num_objects * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_tex_widths, num_objects * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_tex_heights, num_objects * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_output, width * height * 3 * sizeof(unsigned char)));
    CHECK_CUDA(hipMalloc(&d_zbuffer, width * height * sizeof(float)));

    // Copy data to GPU
    int triangle_offset = 0, texture_offset = 0;
    for (int i = 0; i < num_objects; i++) {
        CHECK_CUDA(hipMemcpy(d_triangles + triangle_offset, triangles[i].data(), triangles[i].size() * sizeof(Triangle), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_textures + texture_offset, textures[i], tex_widths[i] * tex_heights[i] * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
        triangle_offset += triangles[i].size();
        texture_offset += tex_widths[i] * tex_heights[i] * 3;
    }

    int triangle_counts[num_objects] = {(int)triangles[0].size(), (int)triangles[1].size()};
    CHECK_CUDA(hipMemcpy(d_triangle_counts, triangle_counts, num_objects * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_tex_widths, tex_widths, num_objects * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_tex_heights, tex_heights, num_objects * sizeof(int), hipMemcpyHostToDevice));

    // Launch kernel
    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

    rasterize_kernel<<<grid_size, block_size>>>(d_triangles, d_triangle_counts, 
                                                d_textures, d_tex_widths, d_tex_heights, 
                                                d_output, d_zbuffer, width, height, num_objects);

    // Copy result back to host and save
    unsigned char* output = new unsigned char[width * height * 3];
    CHECK_CUDA(hipMemcpy(output, d_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));
    stbi_write_png("output.png", width, height, 3, output, width * 3);

    // Clean up
    delete[] output;
    for (int i = 0; i < num_objects; i++) stbi_image_free(textures[i]);
    hipFree(d_triangles);
    hipFree(d_textures);
    hipFree(d_triangle_counts);
    hipFree(d_tex_widths);
    hipFree(d_tex_heights);
    hipFree(d_output);
    hipFree(d_zbuffer);

    return 0;
}