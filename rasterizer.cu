#include "hip/hip_runtime.h"
#include "util.h"

__global__ void render_kernel(Triangle* transformed_triangles, int* triangle_offsets, int* triangle_counts,
                              unsigned char* textures, int* tex_widths, int* tex_heights,
                              unsigned char* output, float* zbuffer,
                              int width, int height, int num_objects, int num_scenes) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int scene = blockIdx.z;
    if (x >= width || y >= height || scene >= num_scenes) return;

    int idx = (scene * height + y) * width + x;
    zbuffer[idx] = FLT_MAX;
    Vec3 color(0.2f, 0.2f, 0.2f);
    Vec3 light_dir(1, 1, 1);
    light_dir = light_dir.normalize();

    for (int obj = 0; obj < num_objects; obj++) {
        int triangle_offset = triangle_offsets[scene * num_objects + obj];
        for (int i = 0; i < triangle_counts[scene * num_objects + obj]; i++) {
            Triangle& tri = transformed_triangles[triangle_offset + i];
            
            Vec3 screen_coords[3];
            for (int j = 0; j < 3; j++) {
                screen_coords[j] = Vec3((tri.v[j].x + 1.0f) * width / 2.0f,
                                        (1.0f - tri.v[j].y) * height / 2.0f,
                                        tri.v[j].z);
            }

            Vec3 edge1 = screen_coords[1] - screen_coords[0];
            Vec3 edge2 = screen_coords[2] - screen_coords[0];
            Vec3 h = Vec3(x, y, 0) - screen_coords[0];
            float det = edge1.x * edge2.y - edge1.y * edge2.x;
            if (fabs(det) < 1e-6) continue;

            float u = (h.x * edge2.y - h.y * edge2.x) / det;
            float v = (edge1.x * h.y - edge1.y * h.x) / det;
            if (u < 0 || v < 0 || u + v > 1) continue;

            float z = screen_coords[0].z + u * (screen_coords[1].z - screen_coords[0].z) +
                      v * (screen_coords[2].z - screen_coords[0].z);
            if (z < zbuffer[idx]) {
                zbuffer[idx] = z;

                Vec2 uv = tri.uv[0] * (1-u-v) + tri.uv[1] * u + tri.uv[2] * v;
                int tex_x = uv.u * tex_widths[scene * num_objects + obj];
                int tex_y = (1.0f - uv.v) * tex_heights[scene * num_objects + obj];
                int tex_idx = (tex_y * tex_widths[scene * num_objects + obj] + tex_x) * 3;
                Vec3 tex_color(textures[tex_idx] / 255.0f,
                               textures[tex_idx + 1] / 255.0f,
                               textures[tex_idx + 2] / 255.0f);

                Vec3 normal = (tri.n[0] * (1-u-v) + tri.n[1] * u + tri.n[2] * v).normalize();
                float diffuse = max(0.0f, normal.dot(light_dir));
                
                color = tex_color * (0.3f + 0.7f * diffuse);
            }
        }
        textures += tex_widths[scene * num_objects + obj] * tex_heights[scene * num_objects + obj] * 3;
    }

    output[idx * 3 + 0] = static_cast<unsigned char>(min(color.x * 255.0f, 255.0f));
    output[idx * 3 + 1] = static_cast<unsigned char>(min(color.y * 255.0f, 255.0f));
    output[idx * 3 + 2] = static_cast<unsigned char>(min(color.z * 255.0f, 255.0f));
}

__global__ void transform_vertices_kernel(Triangle* input_triangles, Triangle* output_triangles, 
                                          int* triangle_offsets, int* triangle_counts, 
                                          Mat4* model_matrices, Mat4 projection, int num_objects, int num_scenes) {
    int scene = blockIdx.y;
    int obj = blockIdx.z;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (scene >= num_scenes || obj >= num_objects) return;
    
    int offset = triangle_offsets[scene * num_objects + obj];
    int count = triangle_counts[scene * num_objects + obj];
    
    if (idx >= count) return;

    Triangle input_tri = input_triangles[offset + idx];
    Triangle& output_tri = output_triangles[offset + idx];
    Mat4 model = model_matrices[scene * num_objects + obj];
    Mat4 mp = projection * model;

    for (int j = 0; j < 3; j++) {
        output_tri.v[j] = mp.multiplyPoint(input_tri.v[j]);
        output_tri.n[j] = model.multiplyVector(input_tri.n[j]).normalize();
        output_tri.uv[j] = input_tri.uv[j];  // Copy UV coordinates
    }
}

// Function to update drone dynamics
void update_drone_dynamics(std::vector<Vec3>& angular_velocity_B, std::vector<Vec3>& linear_velocity_W,
                           std::vector<Vec3>& linear_position_W, std::vector<Mat3>& R_W_B,
                           std::vector<float>& omega, Mat4 model_matrices[][2], float dt) {
    const float k_f = 0.0004905f;
    const float k_m = 0.00004905f;
    const float L = 0.25f;
    const float I[3] = {0.01f, 0.02f, 0.01f};
    const float g = 9.81f;
    const float m = 0.5f;
    const float omega_min = 30.0f;
    const float omega_max = 70.0f;

    for (int scene = 0; scene < angular_velocity_B.size(); scene++) {
        // Limit motor speeds
        for (int i = 0; i < 4; i++) {
            omega[i] = std::max(std::min(omega[i], omega_max), omega_min);
        }

        // Forces and moments
        float F[4], M[4];
        for (int i = 0; i < 4; i++) {
            F[i] = k_f * omega[i] * std::abs(omega[i]);
            M[i] = k_m * omega[i] * std::abs(omega[i]);
        }

        // Thrust
        Vec3 f_B_thrust(0, F[0] + F[1] + F[2] + F[3], 0);

        // Torque
        Vec3 tau_B_drag(0, M[0] - M[1] + M[2] - M[3], 0);
        Vec3 tau_B_thrust = 
            Vec3(-L, 0, L).cross(Vec3(0, F[0], 0)) +
            Vec3(L, 0, L).cross(Vec3(0, F[1], 0)) +
            Vec3(L, 0, -L).cross(Vec3(0, F[2], 0)) +
            Vec3(-L, 0, -L).cross(Vec3(0, F[3], 0));
        Vec3 tau_B = tau_B_drag + tau_B_thrust;

        // Accelerations
        Vec3 linear_acceleration_W = Vec3(0, -g * m, 0) + R_W_B[scene] * f_B_thrust;
        linear_acceleration_W = linear_acceleration_W * (1.0f / m);

        Mat3 I_mat = Mat3::diag(I[0], I[1], I[2]);
        Vec3 angular_acceleration_B = (-angular_velocity_B[scene].cross(I_mat * angular_velocity_B[scene])) + tau_B;
        angular_acceleration_B.x /= I[0];
        angular_acceleration_B.y /= I[1];
        angular_acceleration_B.z /= I[2];

        // Advance state
        linear_velocity_W[scene] += linear_acceleration_W * dt;
        linear_position_W[scene] += linear_velocity_W[scene] * dt;
        angular_velocity_B[scene] += angular_acceleration_B * dt;
        R_W_B[scene] += R_W_B[scene] * skew(angular_velocity_B[scene]) * dt;

        // Update drone model matrix
        model_matrices[scene][1] = Mat4::identity();
        model_matrices[scene][1].setTranslation(linear_position_W[scene]);
        model_matrices[scene][1].setRotation(R_W_B[scene]);
        model_matrices[scene][1] = Mat4::scale(0.01f, 0.01f, 0.01f) * model_matrices[scene][1];
    }
}

int main() {
    const int width = 400, height = 300;
    const int num_objects = 2;
    const int num_scenes = 4;
    const int num_frames = 1000;
    
    std::vector<std::vector<Triangle>> triangles(num_objects);
    std::vector<unsigned char*> textures(num_objects);
    std::vector<int> tex_widths(num_objects), tex_heights(num_objects);

    // Load objects and textures
    load_obj("african_head.obj", triangles[0]);
    load_obj("drone.obj", triangles[1]);
    
    textures[0] = stbi_load("african_head_diffuse.tga", &tex_widths[0], &tex_heights[0], nullptr, 3);
    textures[1] = stbi_load("drone.png", &tex_widths[1], &tex_heights[1], nullptr, 3);
    
    // Prepare projection matrix
    Mat4 projection = create_projection_matrix(3.14159f / 4.0f, (float)width / height, 0.1f, 100.0f);

    // Define initial model matrices for all scenes
    Mat4 model_matrices[num_scenes][num_objects];
    for (int scene = 0; scene < num_scenes; scene++) {
        for (int obj = 0; obj < num_objects; obj++) {
            model_matrices[scene][obj] = create_model_matrix_random();
        }
    }

    std::vector<Triangle> all_triangles;
    std::vector<int> triangle_offsets(num_scenes * num_objects), triangle_counts(num_scenes * num_objects);
    std::vector<unsigned char> all_textures;
    std::vector<int> all_tex_widths(num_scenes * num_objects), all_tex_heights(num_scenes * num_objects);

    for (int scene = 0; scene < num_scenes; scene++) {
        for (int i = 0; i < num_objects; i++) {
            triangle_offsets[scene * num_objects + i] = all_triangles.size();
            triangle_counts[scene * num_objects + i] = triangles[i].size();
            
            all_triangles.insert(all_triangles.end(), triangles[i].begin(), triangles[i].end());
            
            all_tex_widths[scene * num_objects + i] = tex_widths[i];
            all_tex_heights[scene * num_objects + i] = tex_heights[i];
            all_textures.insert(all_textures.end(), textures[i], textures[i] + tex_widths[i] * tex_heights[i] * 3);
        }
    }

    // Allocate GPU memory
    Triangle *d_input_triangles, *d_transformed_triangles;
    int *d_triangle_offsets, *d_triangle_counts, *d_tex_widths, *d_tex_heights;
    unsigned char *d_textures, *d_output;
    float* d_zbuffer;
    Mat4* d_model_matrices;

    CHECK_CUDA(hipMalloc(&d_input_triangles, all_triangles.size() * sizeof(Triangle)));
    CHECK_CUDA(hipMalloc(&d_transformed_triangles, all_triangles.size() * sizeof(Triangle)));
    CHECK_CUDA(hipMalloc(&d_triangle_offsets, num_scenes * num_objects * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_triangle_counts, num_scenes * num_objects * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_textures, all_textures.size() * sizeof(unsigned char)));
    CHECK_CUDA(hipMalloc(&d_tex_widths, num_scenes * num_objects * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_tex_heights, num_scenes * num_objects * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_output, num_scenes * width * height * 3 * sizeof(unsigned char)));
    CHECK_CUDA(hipMalloc(&d_zbuffer, num_scenes * width * height * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_model_matrices, num_scenes * num_objects * sizeof(Mat4)));

    // Copy static data to GPU
    CHECK_CUDA(hipMemcpy(d_input_triangles, all_triangles.data(), all_triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_triangle_offsets, triangle_offsets.data(), num_scenes * num_objects * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_triangle_counts, triangle_counts.data(), num_scenes * num_objects * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_textures, all_textures.data(), all_textures.size() * sizeof(unsigned char), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_tex_widths, all_tex_widths.data(), num_scenes * num_objects * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_tex_heights, all_tex_heights.data(), num_scenes * num_objects * sizeof(int), hipMemcpyHostToDevice));

    // Set up kernel configurations
    int max_triangles = *std::max_element(triangle_counts.begin(), triangle_counts.end());
    dim3 transform_block_size(256);
    dim3 transform_grid_size((max_triangles + transform_block_size.x - 1) / transform_block_size.x, num_scenes, num_objects);
    
    dim3 render_block_size(16, 16, 1);
    dim3 render_grid_size((width + render_block_size.x - 1) / render_block_size.x, 
                          (height + render_block_size.y - 1) / render_block_size.y, 
                          num_scenes);

    const float dt = 0.01f;

    // Prepare video writers for each scene
    std::vector<cv::VideoWriter> video_writers(num_scenes);
    for (int scene = 0; scene < num_scenes; scene++) {
        std::string filename = "output_scene" + std::to_string(scene) + ".mp4";
        video_writers[scene].open(filename, cv::VideoWriter::fourcc('a','v','c','1'), static_cast<int>(std::round(1.0f / dt)), cv::Size(width, height));
        if (!video_writers[scene].isOpened()) {
            std::cerr << "Could not open the output video file for writing: " << filename << std::endl;
            return -1;
        }
    }

    // Drone state variables
    std::vector<float> omega(4, 50.01f);
    std::vector<Vec3> angular_velocity_B(num_scenes, Vec3(0, 0, 0));
    std::vector<Vec3> linear_velocity_W(num_scenes, Vec3(0, 0, 0));
    std::vector<Vec3> linear_position_W(num_scenes);
    std::vector<Mat3> R_W_B(num_scenes);

    // Initialize drone positions and orientations
    for (int scene = 0; scene < num_scenes; scene++) {
        linear_position_W[scene] = Vec3(
            model_matrices[scene][1].m[3],
            model_matrices[scene][1].m[7],
            model_matrices[scene][1].m[11]
        );
        R_W_B[scene] = Mat3::identity();
    }

    // Main rendering loop
    std::vector<unsigned char> output(num_scenes * width * height * 3);
    for (int frame = 0; frame < num_frames; frame++) {
        // Update model matrices (rotate objects)
        for (int scene = 0; scene < num_scenes; scene++) {
            // Update non-drone object (e.g., african head)
            float rotation = 0.1f;
            Mat4 rotation_matrix = Mat4::rotationY(rotation);
            Vec3 translation(model_matrices[scene][0].m[3], 
                             model_matrices[scene][0].m[7], 
                             model_matrices[scene][0].m[11]);
            model_matrices[scene][0] = rotation_matrix * model_matrices[scene][0];
            model_matrices[scene][0].setTranslation(translation);
        }

        // Update drone dynamics
        update_drone_dynamics(angular_velocity_B, linear_velocity_W, linear_position_W, R_W_B, omega, model_matrices, dt);

        // Copy updated model matrices to GPU
        CHECK_CUDA(hipMemcpy(d_model_matrices, model_matrices, num_scenes * num_objects * sizeof(Mat4), hipMemcpyHostToDevice));

        // Transform vertices
        transform_vertices_kernel<<<transform_grid_size, transform_block_size>>>(
            d_input_triangles, d_transformed_triangles, d_triangle_offsets, d_triangle_counts,
            d_model_matrices, projection, num_objects, num_scenes);

        // Render scenes
        render_kernel<<<render_grid_size, render_block_size>>>(
            d_transformed_triangles, d_triangle_offsets, d_triangle_counts,
            d_textures, d_tex_widths, d_tex_heights,
            d_output, d_zbuffer, width, height, num_objects, num_scenes);

        // Copy result back to host
        CHECK_CUDA(hipMemcpy(output.data(), d_output, num_scenes * width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));
        
        // Write frames for each scene
        for (int scene = 0; scene < num_scenes; scene++) {
            cv::Mat frame(height, width, CV_8UC3, output.data() + scene * width * height * 3);
            cv::cvtColor(frame, frame, cv::COLOR_RGB2BGR);  // OpenCV uses BGR by default
            video_writers[scene].write(frame);
        }
    }

    // Close video writers
    for (auto& writer : video_writers) {
        writer.release();
    }

    // Clean up GPU memory
    hipFree(d_input_triangles);
    hipFree(d_transformed_triangles);
    hipFree(d_triangle_offsets);
    hipFree(d_triangle_counts);
    hipFree(d_textures);
    hipFree(d_tex_widths);
    hipFree(d_tex_heights);
    hipFree(d_output);
    hipFree(d_zbuffer);
    hipFree(d_model_matrices);

    // Clean up textures
    for (auto texture : textures) {
        stbi_image_free(texture);
    }

    return 0;
}