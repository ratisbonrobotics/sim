#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <sstream>
#include <string>
#include <algorithm>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <cfloat>
#include <cmath>

#define CHECK_CUDA(call) { hipError_t err = call; if (err != hipSuccess) { printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); exit(1); } }

struct Vec3f {
    float x, y, z;
    __host__ __device__ Vec3f() : x(0), y(0), z(0) {}
    __host__ __device__ Vec3f(float x, float y, float z) : x(x), y(y), z(z) {}
    __host__ __device__ Vec3f operator+(const Vec3f& v) const { return Vec3f(x + v.x, y + v.y, z + v.z); }
    __host__ __device__ Vec3f operator-(const Vec3f& v) const { return Vec3f(x - v.x, y - v.y, z - v.z); }
    __host__ __device__ Vec3f operator*(float f) const { return Vec3f(x * f, y * f, z * f); }
    __host__ __device__ float dot(const Vec3f& v) const { return x * v.x + y * v.y + z * v.z; }
    __host__ __device__ Vec3f cross(const Vec3f& v) const { return Vec3f(y * v.z - z * v.y, z * v.x - x * v.z, x * v.y - y * v.x); }
    __host__ __device__ Vec3f normalize() const { float l = sqrt(x * x + y * y + z * z); return Vec3f(x / l, y / l, z / l); }
    __host__ __device__ float& operator[](int i) { return i == 0 ? x : (i == 1 ? y : z); }
    __host__ __device__ const float& operator[](int i) const { return i == 0 ? x : (i == 1 ? y : z); }
};


struct Vec2f {
    float u, v;
};

struct Triangle {
    Vec3f v[3];
    Vec2f uv[3];
    Vec3f n[3];  // Vertex normals
};

struct Mat4f {
    float m[4][4];

    __host__ __device__ Mat4f() {
        for (int i = 0; i < 4; i++)
            for (int j = 0; j < 4; j++)
                m[i][j] = (i == j) ? 1.0f : 0.0f;
    }

    __host__ __device__ Vec3f transform(const Vec3f& v) const {
        float x = m[0][0] * v.x + m[0][1] * v.y + m[0][2] * v.z + m[0][3];
        float y = m[1][0] * v.x + m[1][1] * v.y + m[1][2] * v.z + m[1][3];
        float z = m[2][0] * v.x + m[2][1] * v.y + m[2][2] * v.z + m[2][3];
        float w = m[3][0] * v.x + m[3][1] * v.y + m[3][2] * v.z + m[3][3];
        return Vec3f(x/w, y/w, z/w);
    }

    __host__ __device__ Vec3f transformNormal(const Vec3f& n) const {
        float x = m[0][0] * n.x + m[0][1] * n.y + m[0][2] * n.z;
        float y = m[1][0] * n.x + m[1][1] * n.y + m[1][2] * n.z;
        float z = m[2][0] * n.x + m[2][1] * n.y + m[2][2] * n.z;
        return Vec3f(x, y, z).normalize();
    }
};

struct Object {
    Triangle* triangles;
    int num_triangles;
    unsigned char* texture;
    int tex_width;
    int tex_height;
    Mat4f model_matrix;
};


__device__ Vec3f barycentric(Vec3f A, Vec3f B, Vec3f C, Vec3f P) {
    Vec3f s[2];
    for (int i = 2; i--; ) {
        s[i].x = C[i] - A[i];
        s[i].y = B[i] - A[i];
        s[i].z = A[i] - P[i];
    }
    Vec3f u = s[0].cross(s[1]);
    if (std::abs(u.z) > 1e-2)
        return Vec3f(1.f - (u.x + u.y) / u.z, u.y / u.z, u.x / u.z);
    return Vec3f(-1, 1, 1);
}

__host__ __device__ Mat4f perspective(float fov, float aspect, float near, float far) {
    Mat4f result;
    float tanHalfFov = tan(fov / 2.0f);
    
    result.m[0][0] = 1.0f / (aspect * tanHalfFov);
    result.m[1][1] = 1.0f / tanHalfFov;
    result.m[2][2] = -(far + near) / (far - near);
    result.m[2][3] = -2.0f * far * near / (far - near);
    result.m[3][2] = -1.0f;
    result.m[3][3] = 0.0f;
    
    return result;
}

__global__ void rasterize_kernel(Object* objects, int num_objects, unsigned char* output, float* zbuffer, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int flipped_y = height - 1 - y;
    int idx = flipped_y * width + x;
    zbuffer[idx] = FLT_MAX;  // Initialize z-buffer to maximum depth

    Vec3f color(0.2f, 0.2f, 0.2f); // Ambient light

    // Create projection matrix
    float fov = 3.14159f / 4.0f;  // 45 degrees
    float aspect = (float)width / height;
    float near = 0.1f;
    float far = 100.0f;
    Mat4f proj = perspective(fov, aspect, near, far);

    for (int obj = 0; obj < num_objects; obj++) {
        Object& object = objects[obj];
        for (int i = 0; i < object.num_triangles; i++) {
            Triangle transformed_triangle = object.triangles[i];
            for (int j = 0; j < 3; j++) {
                // Apply model and projection transformations
                Vec3f v = object.model_matrix.transform(object.triangles[i].v[j]);
                v = proj.transform(v);
                transformed_triangle.v[j] = v;
                
                // Transform normals (only by model matrix, not projection)
                transformed_triangle.n[j] = object.model_matrix.transformNormal(object.triangles[i].n[j]);
            }

            Vec3f screen_coords[3];
            for (int j = 0; j < 3; j++) {
                // Perspective divide and viewport transform
                screen_coords[j] = Vec3f(
                    (transformed_triangle.v[j].x / transformed_triangle.v[j].z + 1.0f) * width / 2.0f,
                    height - 1 - (transformed_triangle.v[j].y / transformed_triangle.v[j].z + 1.0f) * height / 2.0f,
                    transformed_triangle.v[j].z
                );
            }

            Vec3f P(x, flipped_y, 0);
            Vec3f bc_screen = barycentric(screen_coords[0], screen_coords[1], screen_coords[2], P);
            if (bc_screen.x < 0 || bc_screen.y < 0 || bc_screen.z < 0) continue;

            float frag_depth = bc_screen.x * screen_coords[0].z + bc_screen.y * screen_coords[1].z + bc_screen.z * screen_coords[2].z;
            if (frag_depth < zbuffer[idx]) {
                zbuffer[idx] = frag_depth;

                float tex_u = bc_screen.x * object.triangles[i].uv[0].u + bc_screen.y * object.triangles[i].uv[1].u + bc_screen.z * object.triangles[i].uv[2].u;
                float tex_v = bc_screen.x * object.triangles[i].uv[0].v + bc_screen.y * object.triangles[i].uv[1].v + bc_screen.z * object.triangles[i].uv[2].v;

                int tex_x = tex_u * object.tex_width;
                int tex_y = (1.0f - tex_v) * object.tex_height; // Flip V coordinate

                if (tex_x >= 0 && tex_x < object.tex_width && tex_y >= 0 && tex_y < object.tex_height) {
                    Vec3f tex_color;
                    tex_color.x = object.texture[(tex_y * object.tex_width + tex_x) * 3 + 0] / 255.0f;
                    tex_color.y = object.texture[(tex_y * object.tex_width + tex_x) * 3 + 1] / 255.0f;
                    tex_color.z = object.texture[(tex_y * object.tex_width + tex_x) * 3 + 2] / 255.0f;

                    Vec3f normal = (transformed_triangle.n[0] * bc_screen.x +
                                    transformed_triangle.n[1] * bc_screen.y +
                                    transformed_triangle.n[2] * bc_screen.z).normalize();

                    Vec3f light_dir = Vec3f(1, 1, 1).normalize();  // Light direction from top-right-front
                    float diffuse = max(0.0f, normal.dot(light_dir));

                    color = tex_color * (0.3f + 0.7f * diffuse);  // Adjusted ambient and diffuse factors
                }
            }
        }
    }

    output[(flipped_y * width + x) * 3 + 0] = static_cast<unsigned char>(min(color.x * 255.0f, 255.0f));
    output[(flipped_y * width + x) * 3 + 1] = static_cast<unsigned char>(min(color.y * 255.0f, 255.0f));
    output[(flipped_y * width + x) * 3 + 2] = static_cast<unsigned char>(min(color.z * 255.0f, 255.0f));
}

void load_obj(const char* filename, std::vector<Triangle>& triangles) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        printf("Failed to open OBJ file: %s\n", filename);
        return;
    }

    std::vector<Vec3f> vertices;
    std::vector<Vec2f> texcoords;
    std::vector<Vec3f> normals;

    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::string type;
        iss >> type;

        if (type == "v") {
            Vec3f v;
            iss >> v.x >> v.y >> v.z;
            vertices.push_back(v);
        } else if (type == "vt") {
            Vec2f vt;
            iss >> vt.u >> vt.v;
            texcoords.push_back(vt);
        } else if (type == "vn") {
            Vec3f vn;
            iss >> vn.x >> vn.y >> vn.z;
            normals.push_back(vn);
        } else if (type == "f") {
            Triangle tri;
            for (int i = 0; i < 3; i++) {
                int v, vt, vn;
                char slash;
                iss >> v >> slash >> vt >> slash >> vn;
                tri.v[i] = vertices[v - 1];
                tri.uv[i] = texcoords[vt - 1];
                tri.n[i] = normals[vn - 1];
            }
            triangles.push_back(tri);
        }
    }
}

int main() {
    const int width = 800;
    const int height = 600;

    // Load African head
    std::vector<Triangle> african_head_triangles;
    load_obj("african_head.obj", african_head_triangles);
    printf("Loaded African head: %zu triangles\n", african_head_triangles.size());

    int african_head_tex_width, african_head_tex_height, african_head_tex_channels;
    unsigned char* african_head_texture = stbi_load("african_head_diffuse.tga", &african_head_tex_width, &african_head_tex_height, &african_head_tex_channels, 3);
    if (!african_head_texture) {
        printf("Failed to load African head texture\n");
        return 1;
    }
    printf("Loaded African head texture: %dx%d, %d channels\n", african_head_tex_width, african_head_tex_height, african_head_tex_channels);

    // Load drone
    std::vector<Triangle> drone_triangles;
    load_obj("drone.obj", drone_triangles);
    printf("Loaded drone: %zu triangles\n", drone_triangles.size());

    int drone_tex_width, drone_tex_height, drone_tex_channels;
    unsigned char* drone_texture = stbi_load("drone.png", &drone_tex_width, &drone_tex_height, &drone_tex_channels, 3);
    if (!drone_texture) {
        printf("Failed to load drone texture\n");
        return 1;
    }
    printf("Loaded drone texture: %dx%d, %d channels\n", drone_tex_width, drone_tex_height, drone_tex_channels);

    // Prepare objects
    Object objects[2];

    // African head
    objects[0].num_triangles = african_head_triangles.size();
    objects[0].tex_width = african_head_tex_width;
    objects[0].tex_height = african_head_tex_height;

    // African head model matrix
    objects[0].model_matrix.m[0][3] = -1.0f;  // Move left
    objects[0].model_matrix.m[1][3] = 0.0f;
    objects[0].model_matrix.m[2][3] = -3.0f;
    float angle = 3.14159f / 4.0f; // 45 degrees in radians
    float cos_angle = cos(angle);
    float sin_angle = sin(angle);
    objects[0].model_matrix.m[0][0] = cos_angle;
    objects[0].model_matrix.m[0][2] = sin_angle;
    objects[0].model_matrix.m[2][0] = -sin_angle;
    objects[0].model_matrix.m[2][2] = cos_angle;

    // Drone
    objects[1].num_triangles = drone_triangles.size();
    objects[1].tex_width = drone_tex_width;
    objects[1].tex_height = drone_tex_height;

    // Drone model matrix
    objects[1].model_matrix.m[0][3] = 1.0f;   // Move right
    objects[1].model_matrix.m[1][3] = 0.5f;   // Move up
    objects[1].model_matrix.m[2][3] = -2.5f;  // Move closer
    objects[1].model_matrix.m[0][0] = 0.1f;   // Scale down
    objects[1].model_matrix.m[1][1] = 0.1f;
    objects[1].model_matrix.m[2][2] = 0.1f;

    // Allocate memory on device
    CHECK_CUDA(hipMalloc(&objects[0].triangles, african_head_triangles.size() * sizeof(Triangle)));
    CHECK_CUDA(hipMalloc(&objects[0].texture, african_head_tex_width * african_head_tex_height * 3 * sizeof(unsigned char)));
    CHECK_CUDA(hipMalloc(&objects[1].triangles, drone_triangles.size() * sizeof(Triangle)));
    CHECK_CUDA(hipMalloc(&objects[1].texture, drone_tex_width * drone_tex_height * 3 * sizeof(unsigned char)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(objects[0].triangles, african_head_triangles.data(), african_head_triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(objects[0].texture, african_head_texture, african_head_tex_width * african_head_tex_height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(objects[1].triangles, drone_triangles.data(), drone_triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(objects[1].texture, drone_texture, drone_tex_width * drone_tex_height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    // Allocate output buffer and z-buffer
    unsigned char* d_output;
    float* d_zbuffer;
    CHECK_CUDA(hipMalloc(&d_output, width * height * 3 * sizeof(unsigned char)));
    CHECK_CUDA(hipMalloc(&d_zbuffer, width * height * sizeof(float)));
    CHECK_CUDA(hipMemset(d_output, 0, width * height * 3 * sizeof(unsigned char))); // Clear output buffer

    // Copy objects to device
    Object* d_objects;
    CHECK_CUDA(hipMalloc(&d_objects, 2 * sizeof(Object)));
    CHECK_CUDA(hipMemcpy(d_objects, objects, 2 * sizeof(Object), hipMemcpyHostToDevice));

    // Launch kernel
    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

    rasterize_kernel<<<grid_size, block_size>>>(d_objects, 2, d_output, d_zbuffer, width, height);

    // Copy result back to host
    unsigned char* output = new unsigned char[width * height * 3];
    CHECK_CUDA(hipMemcpy(output, d_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Save output image
    stbi_write_png("output.png", width, height, 3, output, width * 3);

    // Clean up
    delete[] output;
    stbi_image_free(african_head_texture);
    stbi_image_free(drone_texture);
    CHECK_CUDA(hipFree(objects[0].triangles));
    CHECK_CUDA(hipFree(objects[0].texture));
    CHECK_CUDA(hipFree(objects[1].triangles));
    CHECK_CUDA(hipFree(objects[1].texture));
    CHECK_CUDA(hipFree(d_output));
    CHECK_CUDA(hipFree(d_zbuffer));
    CHECK_CUDA(hipFree(d_objects));

    return 0;
}