#include "hip/hip_runtime.h"
#include "util.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <algorithm>

__global__ void render_kernel(Triangle* triangles, int* offsets, int* counts,
                              unsigned char* textures, int* tex_widths, int* tex_heights,
                              unsigned char* output, float* zbuffer,
                              int width, int height, int num_objects, int num_scenes) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int scene = blockIdx.z;
    if (x >= width || y >= height || scene >= num_scenes) return;

    int idx = (scene * height + y) * width + x;
    zbuffer[idx] = FLT_MAX;
    Vec3 color(0.2f, 0.2f, 0.2f);
    Vec3 light_dir = Vec3(1, 1, 1).normalize();

    for (int obj = 0; obj < num_objects; obj++) {
        int offset = offsets[scene * num_objects + obj];
        int count = counts[scene * num_objects + obj];
        for (int i = 0; i < count; i++) {
            Triangle& tri = triangles[offset + i];
            Vec3 screen_coords[3];
            for (int j = 0; j < 3; j++) {
                screen_coords[j] = Vec3((tri.v[j].x + 1.0f) * width / 2.0f,
                                        (1.0f - tri.v[j].y) * height / 2.0f,
                                        tri.v[j].z);
            }

            Vec3 edge1 = screen_coords[1] - screen_coords[0];
            Vec3 edge2 = screen_coords[2] - screen_coords[0];
            Vec3 h = Vec3(x, y, 0) - screen_coords[0];
            float det = edge1.x * edge2.y - edge1.y * edge2.x;
            if (fabs(det) < 1e-6) continue;

            float u = (h.x * edge2.y - h.y * edge2.x) / det;
            float v = (edge1.x * h.y - edge1.y * h.x) / det;
            if (u < 0 || v < 0 || u + v > 1) continue;

            float z = screen_coords[0].z + u * (screen_coords[1].z - screen_coords[0].z) +
                      v * (screen_coords[2].z - screen_coords[0].z);
            if (z < zbuffer[idx]) {
                zbuffer[idx] = z;

                Vec2 uv = tri.uv[0] * (1-u-v) + tri.uv[1] * u + tri.uv[2] * v;
                int tex_x = uv.u * tex_widths[scene * num_objects + obj];
                int tex_y = (1.0f - uv.v) * tex_heights[scene * num_objects + obj];
                int tex_idx = (tex_y * tex_widths[scene * num_objects + obj] + tex_x) * 3;
                Vec3 tex_color(textures[tex_idx] / 255.0f,
                               textures[tex_idx + 1] / 255.0f,
                               textures[tex_idx + 2] / 255.0f);

                Vec3 normal = (tri.n[0] * (1-u-v) + tri.n[1] * u + tri.n[2] * v).normalize();
                float diffuse = max(0.0f, normal.dot(light_dir));
                
                color = tex_color * (0.3f + 0.7f * diffuse);
            }
        }
        textures += tex_widths[scene * num_objects + obj] * tex_heights[scene * num_objects + obj] * 3;
    }

    output[idx * 3 + 0] = static_cast<unsigned char>(min(color.x * 255.0f, 255.0f));
    output[idx * 3 + 1] = static_cast<unsigned char>(min(color.y * 255.0f, 255.0f));
    output[idx * 3 + 2] = static_cast<unsigned char>(min(color.z * 255.0f, 255.0f));
}

__global__ void transform_vertices_kernel(Triangle* in, Triangle* out, 
                                          int* offsets, int* counts, 
                                          Mat4* models, Mat4 projection, int num_objects, int num_scenes) {
    int scene = blockIdx.y, obj = blockIdx.z;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (scene >= num_scenes || obj >= num_objects) return;
    
    int offset = offsets[scene * num_objects + obj];
    int count = counts[scene * num_objects + obj];
    if (idx >= count) return;

    Triangle in_tri = in[offset + idx];
    Triangle& out_tri = out[offset + idx];
    Mat4 model = models[scene * num_objects + obj];
    Mat4 mp = projection * model;

    for (int j = 0; j < 3; j++) {
        out_tri.v[j] = mp.multiplyPoint(in_tri.v[j]);
        out_tri.n[j] = model.multiplyVector(in_tri.n[j]).normalize();
        out_tri.uv[j] = in_tri.uv[j];
    }
}

void update_drone_dynamics(std::vector<Vec3>& ang_vel_B, std::vector<Vec3>& lin_vel_W,
                           std::vector<Vec3>& lin_pos_W, std::vector<Mat3>& R_W_B,
                           std::vector<float>& omega, Mat4 model_matrices[][2], float dt) {
    const float k_f = 0.0004905f, k_m = 0.00004905f, L = 0.25f;
    const float I[3] = {0.01f, 0.02f, 0.01f}, g = 9.81f, m = 0.5f;
    const float omega_min = 30.0f, omega_max = 70.0f;

    for (int scene = 0; scene < ang_vel_B.size(); scene++) {
        for (int i = 0; i < 4; i++) omega[i] = std::max(std::min(omega[i], omega_max), omega_min);

        float F[4], M[4];
        for (int i = 0; i < 4; i++) {
            F[i] = k_f * omega[i] * std::abs(omega[i]);
            M[i] = k_m * omega[i] * std::abs(omega[i]);
        }

        Vec3 f_B_thrust(0, F[0] + F[1] + F[2] + F[3], 0);
        Vec3 tau_B_drag(0, M[0] - M[1] + M[2] - M[3], 0);
        Vec3 tau_B_thrust = 
            Vec3(-L, 0, L).cross(Vec3(0, F[0], 0)) +
            Vec3(L, 0, L).cross(Vec3(0, F[1], 0)) +
            Vec3(L, 0, -L).cross(Vec3(0, F[2], 0)) +
            Vec3(-L, 0, -L).cross(Vec3(0, F[3], 0));
        Vec3 tau_B = tau_B_drag + tau_B_thrust;

        Vec3 lin_acc_W = Vec3(0, -g * m, 0) + R_W_B[scene] * f_B_thrust;
        lin_acc_W = lin_acc_W * (1.0f / m);

        Mat3 I_mat = Mat3::diag(I[0], I[1], I[2]);
        Vec3 ang_acc_B = (-ang_vel_B[scene].cross(I_mat * ang_vel_B[scene])) + tau_B;
        ang_acc_B.x /= I[0]; ang_acc_B.y /= I[1]; ang_acc_B.z /= I[2];

        lin_vel_W[scene] += lin_acc_W * dt;
        lin_pos_W[scene] += lin_vel_W[scene] * dt;
        ang_vel_B[scene] += ang_acc_B * dt;
        R_W_B[scene] += R_W_B[scene] * skew(ang_vel_B[scene]) * dt;

        model_matrices[scene][1] = Mat4::identity();
        model_matrices[scene][1].setTranslation(lin_pos_W[scene]);
        model_matrices[scene][1].setRotation(R_W_B[scene]);
        model_matrices[scene][1] = Mat4::scale(0.01f, 0.01f, 0.01f) * model_matrices[scene][1];
    }
}

int main() {
    const int width = 400, height = 300, num_objects = 2, num_scenes = 4, num_frames = 1000;
    
    std::vector<std::vector<Triangle>> triangles(num_objects);
    std::vector<unsigned char*> textures(num_objects);
    std::vector<int> tex_widths(num_objects), tex_heights(num_objects);

    load_obj("african_head.obj", triangles[0]);
    load_obj("drone.obj", triangles[1]);
    
    textures[0] = stbi_load("african_head_diffuse.tga", &tex_widths[0], &tex_heights[0], nullptr, 3);
    textures[1] = stbi_load("drone.png", &tex_widths[1], &tex_heights[1], nullptr, 3);
    
    Mat4 projection = create_projection_matrix(3.14159f / 4.0f, (float)width / height, 0.1f, 100.0f);

    Mat4 model_matrices[num_scenes][num_objects];
    for (int scene = 0; scene < num_scenes; scene++)
        for (int obj = 0; obj < num_objects; obj++)
            model_matrices[scene][obj] = create_model_matrix_random();

    std::vector<Triangle> all_triangles;
    std::vector<int> triangle_offsets(num_scenes * num_objects), triangle_counts(num_scenes * num_objects);
    std::vector<unsigned char> all_textures;
    std::vector<int> all_tex_widths(num_scenes * num_objects), all_tex_heights(num_scenes * num_objects);

    for (int scene = 0; scene < num_scenes; scene++) {
        for (int i = 0; i < num_objects; i++) {
            triangle_offsets[scene * num_objects + i] = all_triangles.size();
            triangle_counts[scene * num_objects + i] = triangles[i].size();
            all_triangles.insert(all_triangles.end(), triangles[i].begin(), triangles[i].end());
            all_tex_widths[scene * num_objects + i] = tex_widths[i];
            all_tex_heights[scene * num_objects + i] = tex_heights[i];
            all_textures.insert(all_textures.end(), textures[i], textures[i] + tex_widths[i] * tex_heights[i] * 3);
        }
    }

    Triangle *d_in_triangles, *d_out_triangles;
    int *d_offsets, *d_counts, *d_tex_widths, *d_tex_heights;
    unsigned char *d_textures, *d_output;
    float* d_zbuffer;
    Mat4* d_model_matrices;

    hipMalloc(&d_in_triangles, all_triangles.size() * sizeof(Triangle));
    hipMalloc(&d_out_triangles, all_triangles.size() * sizeof(Triangle));
    hipMalloc(&d_offsets, num_scenes * num_objects * sizeof(int));
    hipMalloc(&d_counts, num_scenes * num_objects * sizeof(int));
    hipMalloc(&d_textures, all_textures.size() * sizeof(unsigned char));
    hipMalloc(&d_tex_widths, num_scenes * num_objects * sizeof(int));
    hipMalloc(&d_tex_heights, num_scenes * num_objects * sizeof(int));
    hipMalloc(&d_output, num_scenes * width * height * 3 * sizeof(unsigned char));
    hipMalloc(&d_zbuffer, num_scenes * width * height * sizeof(float));
    hipMalloc(&d_model_matrices, num_scenes * num_objects * sizeof(Mat4));

    hipMemcpy(d_in_triangles, all_triangles.data(), all_triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, triangle_offsets.data(), num_scenes * num_objects * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_counts, triangle_counts.data(), num_scenes * num_objects * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_textures, all_textures.data(), all_textures.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_tex_widths, all_tex_widths.data(), num_scenes * num_objects * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_tex_heights, all_tex_heights.data(), num_scenes * num_objects * sizeof(int), hipMemcpyHostToDevice);

    int max_triangles = *std::max_element(triangle_counts.begin(), triangle_counts.end());
    dim3 transform_block(256);
    dim3 transform_grid((max_triangles + transform_block.x - 1) / transform_block.x, num_scenes, num_objects);
    
    dim3 render_block(16, 16, 1);
    dim3 render_grid((width + render_block.x - 1) / render_block.x, 
                     (height + render_block.y - 1) / render_block.y, 
                     num_scenes);

    const float dt = 0.01f;

    std::vector<cv::VideoWriter> video_writers(num_scenes);
    for (int scene = 0; scene < num_scenes; scene++) {
        std::string filename = "output_scene" + std::to_string(scene) + ".mp4";
        video_writers[scene].open(filename, cv::VideoWriter::fourcc('a','v','c','1'), static_cast<int>(std::round(1.0f / dt)), cv::Size(width, height));
    }

    std::vector<float> omega(4, 50.01f);
    std::vector<Vec3> ang_vel_B(num_scenes), lin_vel_W(num_scenes), lin_pos_W(num_scenes);
    std::vector<Mat3> R_W_B(num_scenes, Mat3::identity());

    for (int scene = 0; scene < num_scenes; scene++) {
        lin_pos_W[scene] = Vec3(
            model_matrices[scene][1].m[3],
            model_matrices[scene][1].m[7],
            model_matrices[scene][1].m[11]
        );
    }

    std::vector<unsigned char> output(num_scenes * width * height * 3);
    for (int frame = 0; frame < num_frames; frame++) {
        for (int scene = 0; scene < num_scenes; scene++) {
            float rotation = 0.1f;
            Mat4 rotation_matrix = Mat4::rotationY(rotation);
            Vec3 translation(model_matrices[scene][0].m[3], 
                             model_matrices[scene][0].m[7], 
                             model_matrices[scene][0].m[11]);
            model_matrices[scene][0] = rotation_matrix * model_matrices[scene][0];
            model_matrices[scene][0].setTranslation(translation);
        }

        update_drone_dynamics(ang_vel_B, lin_vel_W, lin_pos_W, R_W_B, omega, model_matrices, dt);

        hipMemcpy(d_model_matrices, model_matrices, num_scenes * num_objects * sizeof(Mat4), hipMemcpyHostToDevice);

        transform_vertices_kernel<<<transform_grid, transform_block>>>(
            d_in_triangles, d_out_triangles, d_offsets, d_counts,
            d_model_matrices, projection, num_objects, num_scenes);

        render_kernel<<<render_grid, render_block>>>(
            d_out_triangles, d_offsets, d_counts,
            d_textures, d_tex_widths, d_tex_heights,
            d_output, d_zbuffer, width, height, num_objects, num_scenes);

        hipMemcpy(output.data(), d_output, num_scenes * width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        
        for (int scene = 0; scene < num_scenes; scene++) {
            cv::Mat frame(height, width, CV_8UC3, output.data() + scene * width * height * 3);
            cv::cvtColor(frame, frame, cv::COLOR_RGB2BGR);
            video_writers[scene].write(frame);
        }
    }

    for (auto& writer : video_writers) {
        writer.release();
    }

    hipFree(d_in_triangles);
    hipFree(d_out_triangles);
    hipFree(d_offsets);
    hipFree(d_counts);
    hipFree(d_textures);
    hipFree(d_tex_widths);
    hipFree(d_tex_heights);
    hipFree(d_output);
    hipFree(d_zbuffer);
    hipFree(d_model_matrices);

    for (auto texture : textures) {
        stbi_image_free(texture);
    }

    return 0;
}