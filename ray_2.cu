
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

class vec3 {
public:
    float e[3];
    __host__ __device__ vec3() {}
    __host__ __device__ vec3(float e0, float e1, float e2) { e[0] = e0; e[1] = e1; e[2] = e2; }
    __host__ __device__ inline float x() const { return e[0]; }
    __host__ __device__ inline float y() const { return e[1]; }
    __host__ __device__ inline float z() const { return e[2]; }
    __host__ __device__ inline float r() const { return e[0]; }
    __host__ __device__ inline float g() const { return e[1]; }
    __host__ __device__ inline float b() const { return e[2]; }

    __host__ __device__ inline const vec3& operator+() const { return *this; }
    __host__ __device__ inline vec3 operator-() const { return vec3(-e[0], -e[1], -e[2]); }
    __host__ __device__ inline float operator[](int i) const { return e[i]; }
    __host__ __device__ inline float& operator[](int i) { return e[i]; }

    __host__ __device__ inline vec3& operator+=(const vec3 &v) { e[0] += v.e[0]; e[1] += v.e[1]; e[2] += v.e[2]; return *this; }
    __host__ __device__ inline vec3& operator*=(const float t) { e[0] *= t; e[1] *= t; e[2] *= t; return *this; }
    __host__ __device__ inline vec3& operator/=(const float t) { return *this *= 1/t; }

    __host__ __device__ inline float length() const { return sqrt(e[0]*e[0] + e[1]*e[1] + e[2]*e[2]); }
    __host__ __device__ inline float squared_length() const { return e[0]*e[0] + e[1]*e[1] + e[2]*e[2]; }
    __host__ __device__ inline void make_unit_vector();
};

__host__ __device__ inline vec3 operator+(const vec3 &v1, const vec3 &v2) { return vec3(v1.e[0] + v2.e[0], v1.e[1] + v2.e[1], v1.e[2] + v2.e[2]); }
__host__ __device__ inline vec3 operator-(const vec3 &v1, const vec3 &v2) { return vec3(v1.e[0] - v2.e[0], v1.e[1] - v2.e[1], v1.e[2] - v2.e[2]); }
__host__ __device__ inline vec3 operator*(const vec3 &v1, const vec3 &v2) { return vec3(v1.e[0] * v2.e[0], v1.e[1] * v2.e[1], v1.e[2] * v2.e[2]); }
__host__ __device__ inline vec3 operator*(float t, const vec3 &v) { return vec3(t*v.e[0], t*v.e[1], t*v.e[2]); }
__host__ __device__ inline vec3 operator*(const vec3 &v, float t) { return t * v; }
__host__ __device__ inline vec3 operator/(vec3 v, float t) { return (1/t) * v; }

__host__ __device__ inline float dot(const vec3 &v1, const vec3 &v2) { return v1.e[0]*v2.e[0] + v1.e[1]*v2.e[1] + v1.e[2]*v2.e[2]; }
__host__ __device__ inline vec3 cross(const vec3 &v1, const vec3 &v2) {
    return vec3(v1.e[1]*v2.e[2] - v1.e[2]*v2.e[1],
                v1.e[2]*v2.e[0] - v1.e[0]*v2.e[2],
                v1.e[0]*v2.e[1] - v1.e[1]*v2.e[0]);
}

__host__ __device__ inline vec3 unit_vector(vec3 v) { return v / v.length(); }

class ray {
public:
    __device__ ray() {}
    __device__ ray(const vec3& a, const vec3& b) { A = a; B = b; }
    __device__ vec3 origin() const { return A; }
    __device__ vec3 direction() const { return B; }
    __device__ vec3 point_at_parameter(float t) const { return A + t*B; }

    vec3 A, B;
};

__device__ vec3 random_in_unit_disk(hiprandState *local_rand_state) {
    vec3 p;
    do {
        p = 2.0f*vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),0) - vec3(1,1,0);
    } while (dot(p,p) >= 1.0f);
    return p;
}

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

class camera {
public:
    __device__ camera(vec3 lookfrom, vec3 lookat, vec3 vup, float vfov, float aspect, float aperture, float focus_dist) {
        lens_radius = aperture / 2.0f;
        float theta = vfov*((float)M_PI)/180.0f;
        float half_height = tan(theta/2.0f);
        float half_width = aspect * half_height;
        origin = lookfrom;
        w = unit_vector(lookfrom - lookat);
        u = unit_vector(cross(vup, w));
        v = cross(w, u);
        lower_left_corner = origin - half_width*focus_dist*u - half_height*focus_dist*v - focus_dist*w;
        horizontal = 2.0f*half_width*focus_dist*u;
        vertical = 2.0f*half_height*focus_dist*v;
    }
    __device__ ray get_ray(float s, float t, hiprandState *local_rand_state) {
        vec3 rd = lens_radius*random_in_unit_disk(local_rand_state);
        vec3 offset = u * rd.x() + v * rd.y();
        return ray(origin + offset, lower_left_corner + s*horizontal + t*vertical - origin - offset);
    }

    vec3 origin;
    vec3 lower_left_corner;
    vec3 horizontal;
    vec3 vertical;
    vec3 u, v, w;
    float lens_radius;
};

class material;

struct hit_record {
    float t;
    vec3 p;
    vec3 normal;
    material *mat_ptr;
};

class hitable {
public:
    __device__ virtual bool hit(const ray& r, float t_min, float t_max, hit_record& rec) const = 0;
};

class sphere: public hitable {
public:
    __device__ sphere() {}
    __device__ sphere(vec3 cen, float r, material *m) : center(cen), radius(r), mat_ptr(m) {};
    __device__ virtual bool hit(const ray& r, float tmin, float tmax, hit_record& rec) const;
    vec3 center;
    float radius;
    material *mat_ptr;
};

__device__ bool sphere::hit(const ray& r, float t_min, float t_max, hit_record& rec) const {
    vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = dot(oc, r.direction());
    float c = dot(oc, oc) - radius*radius;
    float discriminant = b*b - a*c;
    if (discriminant > 0) {
        float temp = (-b - sqrt(discriminant))/a;
        if (temp < t_max && temp > t_min) {
            rec.t = temp;
            rec.p = r.point_at_parameter(rec.t);
            rec.normal = (rec.p - center) / radius;
            rec.mat_ptr = mat_ptr;
            return true;
        }
        temp = (-b + sqrt(discriminant)) / a;
        if (temp < t_max && temp > t_min) {
            rec.t = temp;
            rec.p = r.point_at_parameter(rec.t);
            rec.normal = (rec.p - center) / radius;
            rec.mat_ptr = mat_ptr;
            return true;
        }
    }
    return false;
}

class hitable_list: public hitable {
public:
    __device__ hitable_list() {}
    __device__ hitable_list(hitable **l, int n) { list = l; list_size = n; }
    __device__ virtual bool hit(const ray& r, float tmin, float tmax, hit_record& rec) const;
    hitable **list;
    int list_size;
};

__device__ bool hitable_list::hit(const ray& r, float t_min, float t_max, hit_record& rec) const {
    hit_record temp_rec;
    bool hit_anything = false;
    float closest_so_far = t_max;
    for (int i = 0; i < list_size; i++) {
        if (list[i]->hit(r, t_min, closest_so_far, temp_rec)) {
            hit_anything = true;
            closest_so_far = temp_rec.t;
            rec = temp_rec;
        }
    }
    return hit_anything;
}

#define RANDVEC3 vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))

__device__ vec3 random_in_unit_sphere(hiprandState *local_rand_state) {
    vec3 p;
    do {
        p = 2.0f*RANDVEC3 - vec3(1,1,1);
    } while (p.squared_length() >= 1.0f);
    return p;
}

__device__ vec3 reflect(const vec3& v, const vec3& n) {
     return v - 2.0f*dot(v,n)*n;
}

class material {
public:
    __device__ virtual bool scatter(const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState *local_rand_state) const = 0;
};

class lambertian : public material {
public:
    __device__ lambertian(const vec3& a) : albedo(a) {}
    __device__ virtual bool scatter(const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState *local_rand_state) const {
        vec3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
        scattered = ray(rec.p, target-rec.p);
        attenuation = albedo;
        return true;
    }

    vec3 albedo;
};

class metal : public material {
public:
    __device__ metal(const vec3& a, float f) : albedo(a) { if (f < 1) fuzz = f; else fuzz = 1; }
    __device__ virtual bool scatter(const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState *local_rand_state) const {
        vec3 reflected = reflect(unit_vector(r_in.direction()), rec.normal);
        scattered = ray(rec.p, reflected + fuzz*random_in_unit_sphere(local_rand_state));
        attenuation = albedo;
        return (dot(scattered.direction(), rec.normal) > 0.0f);
    }
    vec3 albedo;
    float fuzz;
};

__device__ float schlick(float cosine, float ref_idx) {
    float r0 = (1.0f-ref_idx) / (1.0f+ref_idx);
    r0 = r0*r0;
    return r0 + (1.0f-r0)*pow((1.0f - cosine),5.0f);
}

__device__ bool refract(const vec3& v, const vec3& n, float ni_over_nt, vec3& refracted) {
    vec3 uv = unit_vector(v);
    float dt = dot(uv, n);
    float discriminant = 1.0f - ni_over_nt*ni_over_nt*(1-dt*dt);
    if (discriminant > 0) {
        refracted = ni_over_nt*(uv - n*dt) - n*sqrt(discriminant);
        return true;
    }
    else
        return false;
}

class dielectric : public material {
public:
    __device__ dielectric(float ri) : ref_idx(ri) {}
    __device__ virtual bool scatter(const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState *local_rand_state) const {
        vec3 outward_normal;
        vec3 reflected = reflect(r_in.direction(), rec.normal);
        float ni_over_nt;
        attenuation = vec3(1.0, 1.0, 1.0);
        vec3 refracted;
        float reflect_prob;
        float cosine;
        if (dot(r_in.direction(), rec.normal) > 0.0f) {
            outward_normal = -rec.normal;
            ni_over_nt = ref_idx;
            cosine = dot(r_in.direction(), rec.normal) / r_in.direction().length();
            cosine = sqrt(1.0f - ref_idx*ref_idx*(1-cosine*cosine));
        }
        else {
            outward_normal = rec.normal;
            ni_over_nt = 1.0f / ref_idx;
            cosine = -dot(r_in.direction(), rec.normal) / r_in.direction().length();
        }
        if (refract(r_in.direction(), outward_normal, ni_over_nt, refracted))
            reflect_prob = schlick(cosine, ref_idx);
        else
            reflect_prob = 1.0f;
        if (hiprand_uniform(local_rand_state) < reflect_prob)
            scattered = ray(rec.p, reflected);
        else
            scattered = ray(rec.p, refracted);
        return true;
    }

    float ref_idx;
};

__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation = cur_attenuation * attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]); col[1] = sqrt(col[1]); col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000,0), 1000, new lambertian(vec3(0.5,0.5,0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2, new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                } else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                        new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                } else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, 22*22+1+3);

        vec3 lookfrom(13,2,3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0;
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom, lookat, vec3(0,1,0), 20.0, float(nx)/float(ny), aperture, dist_to_focus);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main() {
    int nx = 1200;
    int ny = 800;
    int ns = 10;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    int num_hitables = 22*22+1+3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
    return 0;
}