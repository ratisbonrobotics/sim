#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <sstream>
#include <string>
#include <algorithm>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <cfloat>
#include <cmath>

#define CHECK_CUDA(call) { hipError_t err = call; if (err != hipSuccess) { printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); exit(1); } }

struct Vec3f {
    float x, y, z;
    __host__ __device__ Vec3f() : x(0), y(0), z(0) {}
    __host__ __device__ Vec3f(float x, float y, float z) : x(x), y(y), z(z) {}
    __host__ __device__ Vec3f operator+(const Vec3f& v) const { return Vec3f(x + v.x, y + v.y, z + v.z); }
    __host__ __device__ Vec3f operator-(const Vec3f& v) const { return Vec3f(x - v.x, y - v.y, z - v.z); }
    __host__ __device__ Vec3f operator*(float f) const { return Vec3f(x * f, y * f, z * f); }
    __host__ __device__ float dot(const Vec3f& v) const { return x * v.x + y * v.y + z * v.z; }
    __host__ __device__ Vec3f cross(const Vec3f& v) const { return Vec3f(y * v.z - z * v.y, z * v.x - x * v.z, x * v.y - y * v.x); }
    __host__ __device__ Vec3f normalize() const { float l = sqrt(x * x + y * y + z * z); return Vec3f(x / l, y / l, z / l); }
};

struct Vec2f {
    float u, v;
};

struct Triangle {
    Vec3f v[3];
    Vec2f uv[3];
    Vec3f n[3];  // Vertex normals
};

struct Ray {
    Vec3f origin;
    Vec3f direction;
};

struct Mat4f {
    float m[4][4];

    __host__ __device__ Mat4f() {
        for (int i = 0; i < 4; i++)
            for (int j = 0; j < 4; j++)
                m[i][j] = (i == j) ? 1.0f : 0.0f;
    }

    __host__ __device__ Vec3f transform(const Vec3f& v) const {
        float x = m[0][0] * v.x + m[0][1] * v.y + m[0][2] * v.z + m[0][3];
        float y = m[1][0] * v.x + m[1][1] * v.y + m[1][2] * v.z + m[1][3];
        float z = m[2][0] * v.x + m[2][1] * v.y + m[2][2] * v.z + m[2][3];
        float w = m[3][0] * v.x + m[3][1] * v.y + m[3][2] * v.z + m[3][3];
        return Vec3f(x/w, y/w, z/w);
    }

    __host__ __device__ Vec3f transformNormal(const Vec3f& n) const {
        float x = m[0][0] * n.x + m[0][1] * n.y + m[0][2] * n.z;
        float y = m[1][0] * n.x + m[1][1] * n.y + m[1][2] * n.z;
        float z = m[2][0] * n.x + m[2][1] * n.y + m[2][2] * n.z;
        return Vec3f(x, y, z).normalize();
    }
};

__device__ bool ray_triangle_intersect(const Ray& ray, const Triangle& triangle, float& t, float& u, float& v) {
    Vec3f edge1 = triangle.v[1] - triangle.v[0];
    Vec3f edge2 = triangle.v[2] - triangle.v[0];
    Vec3f h = ray.direction.cross(edge2);
    float a = edge1.dot(h);

    if (a > -1e-5 && a < 1e-5) return false;

    float f = 1.0f / a;
    Vec3f s = ray.origin - triangle.v[0];
    u = f * s.dot(h);

    if (u < 0.0f || u > 1.0f) return false;

    Vec3f q = s.cross(edge1);
    v = f * ray.direction.dot(q);

    if (v < 0.0f || u + v > 1.0f) return false;

    t = f * edge2.dot(q);

    return t > 1e-5;
}

__global__ void ray_trace_kernel(Triangle* triangles, int num_triangles, unsigned char* texture, int tex_width, int tex_height, unsigned char* output, int width, int height, Mat4f model_matrix) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float aspect_ratio = (float)width / height;
    float fov = 3.14159f / 4.0f;
    float tan_fov = tan(fov / 2.0f);

    float camera_x = ((2.0f * (x + 0.5f) / width - 1.0f) * aspect_ratio) * tan_fov;
    float camera_y = (1.0f - 2.0f * (y + 0.5f) / height) * tan_fov;

    Ray ray;
    ray.origin = Vec3f(0, 0, 3);  // Move camera to positive z-axis
    ray.direction = Vec3f(camera_x, camera_y, -1).normalize();  // Look towards negative z-axis

    Vec3f color(0.2f, 0.2f, 0.2f); // Ambient light
    float closest_t = FLT_MAX;

    for (int i = 0; i < num_triangles; i++) {
        Triangle transformed_triangle = triangles[i];
        for (int j = 0; j < 3; j++) {
            transformed_triangle.v[j] = model_matrix.transform(triangles[i].v[j]);
            transformed_triangle.n[j] = model_matrix.transformNormal(triangles[i].n[j]);
        }

        float t, u, v;
        if (ray_triangle_intersect(ray, transformed_triangle, t, u, v) && t < closest_t) {
            closest_t = t;

            // Barycentric coordinates
            float w = 1.0f - u - v;

            // Texture coordinates
            float tex_u = w * triangles[i].uv[0].u + u * triangles[i].uv[1].u + v * triangles[i].uv[2].u;
            float tex_v = w * triangles[i].uv[0].v + u * triangles[i].uv[1].v + v * triangles[i].uv[2].v;

            int tex_x = tex_u * tex_width;
            int tex_y = (1.0f - tex_v) * tex_height; // Flip V coordinate

            if (tex_x >= 0 && tex_x < tex_width && tex_y >= 0 && tex_y < tex_height) {
                Vec3f tex_color;
                tex_color.x = texture[(tex_y * tex_width + tex_x) * 3 + 0] / 255.0f;
                tex_color.y = texture[(tex_y * tex_width + tex_x) * 3 + 1] / 255.0f;
                tex_color.z = texture[(tex_y * tex_width + tex_x) * 3 + 2] / 255.0f;

                // Interpolate vertex normals
                Vec3f normal = (transformed_triangle.n[0] * w + transformed_triangle.n[1] * u + transformed_triangle.n[2] * v).normalize();

                Vec3f light_dir = Vec3f(1, 1, 1).normalize();  // Light direction from top-right-front
                float diffuse = max(0.0f, normal.dot(light_dir));

                color = tex_color * (0.3f + 0.7f * diffuse);  // Adjusted ambient and diffuse factors
            }
        }
    }

    output[(y * width + x) * 3 + 0] = static_cast<unsigned char>(min(color.x * 255.0f, 255.0f));
    output[(y * width + x) * 3 + 1] = static_cast<unsigned char>(min(color.y * 255.0f, 255.0f));
    output[(y * width + x) * 3 + 2] = static_cast<unsigned char>(min(color.z * 255.0f, 255.0f));
}

void load_obj(const char* filename, std::vector<Triangle>& triangles) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        printf("Failed to open OBJ file\n");
        return;
    }

    std::vector<Vec3f> vertices;
    std::vector<Vec2f> texcoords;
    std::vector<Vec3f> normals;

    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::string type;
        iss >> type;

        if (type == "v") {
            Vec3f v;
            iss >> v.x >> v.y >> v.z;
            vertices.push_back(v);
        } else if (type == "vt") {
            Vec2f vt;
            iss >> vt.u >> vt.v;
            texcoords.push_back(vt);
        } else if (type == "vn") {
            Vec3f vn;
            iss >> vn.x >> vn.y >> vn.z;
            normals.push_back(vn);
        } else if (type == "f") {
            Triangle tri;
            for (int i = 0; i < 3; i++) {
                int v, vt, vn;
                char slash;
                iss >> v >> slash >> vt >> slash >> vn;
                tri.v[i] = vertices[v - 1];
                tri.uv[i] = texcoords[vt - 1];
                tri.n[i] = normals[vn - 1];
            }
            triangles.push_back(tri);
        }
    }
}

int main() {
    const int width = 800;
    const int height = 600;

    std::vector<Triangle> triangles;
    load_obj("african_head.obj", triangles);

    printf("Loaded %zu triangles\n", triangles.size());

    int tex_width, tex_height, tex_channels;
    unsigned char* texture = stbi_load("african_head_diffuse.tga", &tex_width, &tex_height, &tex_channels, 3);
    if (!texture) {
        printf("Failed to load texture\n");
        return 1;
    }
    printf("Loaded texture: %dx%d, %d channels\n", tex_width, tex_height, tex_channels);

    Triangle* d_triangles;
    unsigned char* d_texture;
    unsigned char* d_output;

    CHECK_CUDA(hipMalloc(&d_triangles, triangles.size() * sizeof(Triangle)));
    CHECK_CUDA(hipMalloc(&d_texture, tex_width * tex_height * 3 * sizeof(unsigned char)));
    CHECK_CUDA(hipMalloc(&d_output, width * height * 3 * sizeof(unsigned char)));

    CHECK_CUDA(hipMemcpy(d_triangles, triangles.data(), triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_texture, texture, tex_width * tex_height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_output, 0, width * height * 3 * sizeof(unsigned char))); // Clear output buffer

    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

    // Create model matrix for translation and rotation
    Mat4f model_matrix;
    // Translate the model
    model_matrix.m[0][3] = 0.0f;  // Move 0 units along x-axis
    model_matrix.m[1][3] = 0.0f; // Move 0 units along y-axis
    model_matrix.m[2][3] = -3.0f;  // No movement along z-axis

    // Rotate the model (example: rotate 45 degrees around y-axis)
    float angle = 3.14159f / 4.0f; // 45 degrees in radians
    float cos_angle = cos(angle);
    float sin_angle = sin(angle);
    model_matrix.m[0][0] = cos_angle;
    model_matrix.m[0][2] = sin_angle;
    model_matrix.m[2][0] = -sin_angle;
    model_matrix.m[2][2] = cos_angle;

    ray_trace_kernel<<<grid_size, block_size>>>(d_triangles, triangles.size(), d_texture, tex_width, tex_height, d_output, width, height, model_matrix);

    unsigned char* output = new unsigned char[width * height * 3];
    CHECK_CUDA(hipMemcpy(output, d_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

    stbi_write_png("output.png", width, height, 3, output, width * 3);

    delete[] output;
    stbi_image_free(texture);
    CHECK_CUDA(hipFree(d_triangles));
    CHECK_CUDA(hipFree(d_texture));
    CHECK_CUDA(hipFree(d_output));

    return 0;
}