#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int size = 1024;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    // Allocate memory on host
    a = (int*)malloc(size * sizeof(int));
    b = (int*)malloc(size * sizeof(int));
    c = (int*)malloc(size * sizeof(int));

    // Initialize input vectors
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate memory on device
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    // Copy input vectors from host to device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 1 block and 1024 threads
    int threadsPerBlock = 1024;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // Copy result vector from device to host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the first few elements of the result vector
    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}