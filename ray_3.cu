#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}

struct Vec3 {
    float x, y, z;
    __host__ __device__ Vec3() {}
    __host__ __device__ Vec3(float x, float y, float z) : x(x), y(y), z(z) {}
    __host__ __device__ Vec3 operator+(const Vec3& v) const { return Vec3(x + v.x, y + v.y, z + v.z); }
    __host__ __device__ Vec3 operator-(const Vec3& v) const { return Vec3(x - v.x, y - v.y, z - v.z); }
    __host__ __device__ Vec3 operator*(float d) const { return Vec3(x * d, y * d, z * d); }
    __host__ __device__ Vec3 operator*(const Vec3& v) const { return Vec3(x * v.x, y * v.y, z * v.z); }
    __host__ __device__ Vec3 normalize() const {
        float mg = sqrtf(x*x + y*y + z*z);
        return Vec3(x/mg, y/mg, z/mg);
    }
};

__host__ __device__ float dot(const Vec3& a, const Vec3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

struct Ray {
    Vec3 origin, direction;
    __host__ __device__ Ray(const Vec3& origin, const Vec3& direction) : origin(origin), direction(direction) {}
};

struct Sphere {
    Vec3 center;
    float radius;
    Vec3 color;
    __host__ __device__ Sphere(const Vec3& center, float radius, const Vec3& color) 
        : center(center), radius(radius), color(color) {}
    
    __host__ __device__ bool intersect(const Ray& ray, float& t) const {
        Vec3 oc = ray.origin - center;
        float a = dot(ray.direction, ray.direction);
        float b = 2.0f * dot(oc, ray.direction);
        float c = dot(oc, oc) - radius * radius;
        float discriminant = b * b - 4 * a * c;
        if (discriminant < 0) return false;
        t = (-b - sqrtf(discriminant)) / (2.0f * a);
        return t > 0;
    }
};

__device__ Vec3 color(const Ray& ray, Sphere* spheres, int sphere_count) {
    float closest_t = INFINITY;
    Sphere* hit_sphere = nullptr;

    for (int i = 0; i < sphere_count; i++) {
        float t;
        if (spheres[i].intersect(ray, t) && t < closest_t) {
            closest_t = t;
            hit_sphere = &spheres[i];
        }
    }

    if (hit_sphere) {
        Vec3 hit_point = ray.origin + ray.direction * closest_t;
        Vec3 normal = (hit_point - hit_sphere->center).normalize();
        Vec3 light_dir = Vec3(1, 1, -1).normalize();
        float diffuse = fmaxf(0.0f, dot(normal, light_dir));
        
        Ray shadow_ray(hit_point + normal * 0.001f, light_dir);
        for (int i = 0; i < sphere_count; i++) {
            float t;
            if (spheres[i].intersect(shadow_ray, t)) {
                diffuse *= 0.5f;
                break;
            }
        }
        
        return hit_sphere->color * (diffuse * 0.7f + 0.2f);
    }

    Vec3 unit_direction = ray.direction.normalize();
    float t = 0.5f * (unit_direction.y + 1.0f);
    return Vec3(1.0f, 1.0f, 1.0f) * (1.0f - t) + Vec3(0.5f, 0.7f, 1.0f) * t;
}

__global__ void render(Vec3* fb, int width, int height, int samples, Sphere* spheres, int sphere_count, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int render_index = blockIdx.z;
    if ((i >= width) || (j >= height)) return;
    int pixel_index = (render_index * width * height) + (j * width + i);
    hiprandState local_rand_state = rand_state[pixel_index];

    Vec3 lower_left_corner(-2.0f, -1.0f, -1.0f);
    Vec3 horizontal(4.0f, 0.0f, 0.0f);
    Vec3 vertical(0.0f, 2.0f, 0.0f);
    Vec3 origin(0.0f, 0.0f, 0.0f);

    Vec3 col(0, 0, 0);
    for (int s = 0; s < samples; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(width);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(height);
        Ray r(origin, lower_left_corner + horizontal * u + vertical * v);
        col = col + color(r, spheres + render_index * sphere_count, sphere_count);
    }
    col = col * (1.0f / float(samples));
    fb[pixel_index] = col;
}

int main() {
    int width = 100;
    int height = 50;
    int samples = 4;
    int sphere_count = 2;

    // Array of scene counts to test
    int scene_counts[] = {2, 4, 8, 16, 32, 64, 128, 256, 512};
    int num_tests = sizeof(scene_counts) / sizeof(scene_counts[0]);

    for (int test = 0; test < num_tests; test++) {
        int num_renders = scene_counts[test];

        Vec3* fb;
        CHECK_CUDA(hipMallocManaged(&fb, num_renders * width * height * sizeof(Vec3)));

        Sphere* spheres;
        CHECK_CUDA(hipMallocManaged(&spheres, num_renders * sphere_count * sizeof(Sphere)));

        srand(time(NULL));
        for (int r = 0; r < num_renders; r++) {
            float x = (float)rand() / RAND_MAX * 2 - 1;  // Random x between -1 and 1
            float y = (float)rand() / RAND_MAX * 2 - 1;  // Random y between -1 and 1
            spheres[r * sphere_count] = Sphere(Vec3(x, y, -1), 0.5f, Vec3(0.7f, 0.3f, 0.3f));
            spheres[r * sphere_count + 1] = Sphere(Vec3(0, -100.5f, -1), 100.0f, Vec3(0.3f, 0.7f, 0.3f));
        }

        hiprandState* rand_state;
        CHECK_CUDA(hipMalloc(&rand_state, num_renders * width * height * sizeof(hiprandState)));

        dim3 blocks(width/16+1, height/16+1, num_renders);
        dim3 threads(16, 16);

        // Measure execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        render<<<blocks, threads>>>(fb, width, height, samples, spheres, sphere_count, rand_state);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipDeviceSynchronize());
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        printf("Number of scenes: %d, Execution time: %.2f ms\n", num_renders, milliseconds);

        CHECK_CUDA(hipFree(fb));
        CHECK_CUDA(hipFree(spheres));
        CHECK_CUDA(hipFree(rand_state));

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    return 0;
}